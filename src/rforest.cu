#include "hip/hip_runtime.h"
/* 
 * Random Forest
 * Vaibhav Anand, 2018
 */

#include "rforest.cuh"

#include <cstdio>
#include <hip/hip_runtime.h>

#include "cuda_header.cuh"
#include "utils.hpp"

#define THREADS_PER_BLOCK 1024


 __global__
 void cuda_get_losses(const float *gpu_in_x, const float *gpu_in_y, 
        float *gpu_tmp, float *gpu_out_x, int num_features, int num_points) {
    extern __shared__ float shmem[];

    unsigned tid = threadIdx.x;
    float part1_n, part1_y, part2_n, part2_y;

    for (uint p = 0; p < num_points; p++) {
        uint i = (blockIdx.x * num_points) + tid; // match to every element in gpu_in_x

        // BLOCKSPECIFIC:
        uint j = (blockIdx.x * num_points) + p; // match to first point in every feature of gpu_in

        // ITERATE by THREADS_PER_BLOCK until we are done with num_points!

        if (tid < num_points) {
            gpu_tmp[i] = (gpu_in_x[i] >= gpu_in_x[j]);
            shmem[4 * tid] = gpu_tmp[i];
            shmem[4 * tid + 1] = shmem[4 * tid] * gpu_in_y[tid];
            shmem[4 * tid + 2] = (1 - shmem[4 * tid]) * gpu_in_y[tid];
            shmem[4 * tid + 3] = 0.;
            // atomicAdd(&shmem[0], gpu_tmp[k]);
        }

        __syncthreads();
        shmem[tid] += shmem[tid + blockDim.x * 3];

        for (uint s = blockDim.x * 2; s > 2; s >>= 1) {
            __syncthreads();
            if (tid < s) {
                shmem[tid] += shmem[tid + s];
                shmem[tid + s] = 0.; // for subsequent kernel runs
            }
        }

        __syncthreads();
        if (tid == 0) {
            part1_n = shmem[0];
            part2_n = num_points - part1_n;
            part1_y = shmem[1] / (part1_n + SMALL);
            part2_y = shmem[2] / (part2_n + SMALL);
            float part1_p = part1_n / num_points;
            float result = GINI(part1_y) * part1_p + GINI(part2_y) * (1 - part1_p);
            // purposely done w/blockIdx.x as row-indexer
            gpu_out_x[blockIdx.x * num_points + p] = result;
        }
    }
}



// num_features does not include y
void cuda_call_get_losses(float *gpu_in_x, float *gpu_in_y, float *gpu_tmp,
    float *gpu_out_x, int num_features, int num_points) {
    // TODO: constraint w/shared memory
    // change to non-hard-coded variables
    cuda_get_losses<<<num_features, THREADS_PER_BLOCK, 
        (4 * THREADS_PER_BLOCK * sizeof(float))>>>(
        gpu_in_x, gpu_in_y, gpu_tmp, gpu_out_x, num_features, num_points);
}

